#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include "stdio.h"
//#include <cuda/cuda_runtime.h>
#include "hipfft/hipfft.h"
    static const char *_cudaGetErrorEnum(hipfftResult error)
    {
    switch (error)
    {
    case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

    case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

    case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

    case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

    case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

    case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

    case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

    case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

    case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

    case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";

    /*case CUFFT_INCOMPLETE_PARAMETER_LIST:
    return "CUFFT_INCOMPLETE_PARAMETER_LIST";

    case CUFFT_INVALID_DEVICE:
    return "CUFFT_INVALID_DEVICE";

    case CUFFT_PARSE_ERROR:
    return "CUFFT_PARSE_ERROR";

    case CUFFT_NO_WORKSPACE:
    return "CUFFT_NO_WORKSPACE";

    case CUFFT_NOT_IMPLEMENTED:
    return "CUFFT_NOT_IMPLEMENTED";

    case CUFFT_NOT_SUPPORTED:
    return "CUFFT_NOT_SUPPORTED";*/
    }

    return "<unknown>";
    }

    inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
    {
    if( HIPFFT_SUCCESS != err) {
    fprintf(stderr, "CUFFT error at 1\n");
    fprintf(stderr, "CUFFT error in file '%s'\n",__FILE__);
    fprintf(stderr, "CUFFT error at 2\n");
    /*fprintf(stderr, "CUFFT error line '%s'\n",__LINE__);*/
    fprintf(stderr, "CUFFT error at 3\n");
    /*fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
    _cudaGetErrorEnum(err)); \*/
    fprintf(stderr, "CUFFT error %d: %s\nterminating!\n",err,_cudaGetErrorEnum(err)); \
    hipDeviceReset(); return; \
    }
    }


static int allocatedWorkspace=0;
static void* planWorkspace;
static int planWorkspaceSize=100*1024*1024; //100MB
 
extern "C"
void
create_plan_ffth_(hipfftHandle * *plan, int *ISIGNp, int *Np, int *LOTp)
{
int ISIGN = *ISIGNp;
int N = *Np;
int LOT = *LOTp;

*plan = new hipfftHandle;
//cufftHandle plan;

if (hipDeviceSynchronize() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}


// //create a single re-usable workspace
// if(!allocatedWorkspace){
//   allocatedWorkspace=1;
//   //allocate plan workspace
//   cudaMalloc(&planWorkspace,planWorkspaceSize);
// }
//
// //disable auto allocation so we can re-use a single workspace (created above)
//  cufftSetAutoAllocation(plan, false);

int embed[1];
int stride;
int dist;

#ifdef TRANS_SINGLE
cufftType cufft_1 = CUFFT_R2C;
cufftType cufft_2 = CUFFT_C2R;
#else
hipfftType cufft_1 = HIPFFT_D2Z;
hipfftType cufft_2 = HIPFFT_Z2D;
#endif

embed[0] = 1;
stride   = LOT;
dist     = 1;

cufftSafeCall(hipfftCreate(*plan));

//printf("CreatePlan cufft\n","N=",N);
//printf("%s %d \n","plan=",plan);
//printf("%s %d \n","LOT=",LOT);
//printf("%s %d \n","ISIGN=",ISIGN);
//printf("%s %d \n","Np=",*Np);

if( ISIGN== -1 ){
  cufftSafeCall(hipfftPlanMany(*plan, 1, &N,
                 embed, stride, dist, 
                 embed, stride, dist, 
                 cufft_1, LOT));
  //cufftSafeCall(cufftPlan1d(&plan, N, CUFFT_D2Z, LOT));
}
else if( ISIGN== 1){
  cufftSafeCall(hipfftPlanMany(*plan, 1, &N,
                 embed, stride, dist, 
                 embed, stride, dist, 
                 cufft_2, LOT));
  //cufftSafeCall(cufftPlan1d(&plan, N, CUFFT_Z2D, LOT));
}
else {
  abort();
}

// // use our reusaable work area for the plan
// cufftSetWorkArea(plan,planWorkspace); 

/*
if( ISIGN== -1 ){
  cufftSafeCall(cufftPlan1d(&plan, N, CUFFT_D2Z, LOT));
}
else if( ISIGN== 1){
  cufftSafeCall(cufftPlan1d(&plan, N, CUFFT_Z2D, LOT));
}
else {
  abort();
}
*/

if (hipDeviceSynchronize() != hipSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}

//*PLANp=plan;
//fprintf(stderr, "create_plan_ffth_: plan-address = %p\n",*plan);

// // get size used by this plan
// size_t workSize;
// cufftGetSize(plan,&workSize);
//
// // exit if we don't have enough space for the work area in the re-usable workspace
// if(workSize > planWorkspaceSize){
//   printf("create_plan_ffth: plan workspace size not large enough - exiting\n");
// exit(1);
// }


return;


}

